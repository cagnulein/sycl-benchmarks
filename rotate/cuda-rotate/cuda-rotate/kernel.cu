#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "../../../includes/dimensions.h"
#include <stdio.h>
#include <math.h>

float cosine;
float sine;

float Point1x;
float Point1y;
float Point2x;
float Point2y;
float Point3x;
float Point3y;

float minx;
float miny;
float maxx;
float maxy;

long DestBitmapWidth;
long DestBitmapHeight;


hipError_t rotateWithCuda(unsigned char *src, unsigned char *dest, unsigned int src_size, unsigned int dest_size);

__global__ void rotateKernel(unsigned char* src, unsigned char* dest, long DestBitmapWidth, long DestBitmapHeight, float sine, float cosine, float minx, float miny)
{
	int i = threadIdx.x;
	int x = i / DestBitmapWidth;
	int y = i % DestBitmapHeight;

	int SrcBitmapx = ((x + minx)*cosine + (y + miny)*sine);
	int SrcBitmapy = ((y + miny)*cosine - (x + minx)*sine);
	if (SrcBitmapx >= 0 && SrcBitmapx < IMAGE_WIDTH && SrcBitmapy >= 0 && SrcBitmapy < IMAGE_HEIGHT)
		dest[(x * DestBitmapWidth) + y] = src[(SrcBitmapx * IMAGE_WIDTH) + SrcBitmapy];
}

unsigned char* old_image;
unsigned char* new_image;

int main(int argc, char *argv[])
{
	old_image = new unsigned char[IMAGE_SIZE];

	double degrees = 180;
	if (argc > 1)
	{
		degrees = atof(argv[1]);
		//printf("%f degrees forced\n", degrees);
	}

	float radians = (2 * 3.1416*degrees) / 360;

	cosine = (float)cos(radians);
	sine = (float)sin(radians);

	Point1x = (-IMAGE_HEIGHT * sine);
	Point1y = (IMAGE_HEIGHT*cosine);
	Point2x = (IMAGE_WIDTH*cosine - IMAGE_HEIGHT * sine);
	Point2y = (IMAGE_HEIGHT*cosine + IMAGE_WIDTH * sine);
	Point3x = (IMAGE_WIDTH*cosine);
	Point3y = (IMAGE_WIDTH*sine);

	minx = min((float)0.0, min(Point1x, min(Point2x, Point3x)));
	miny = min((float)0.0, min(Point1y, min(Point2y, Point3y)));
	maxx = max(Point1x, max(Point2x, Point3x));
	maxy = max(Point1y, max(Point2y, Point3y));

	DestBitmapWidth = (int)ceil(fabs(maxx) - minx);
	DestBitmapHeight = (int)ceil(fabs(maxy) - miny);

	new_image = new unsigned char[DestBitmapWidth*DestBitmapHeight];

	// Add vectors in parallel.
	hipError_t cudaStatus = rotateWithCuda(old_image, new_image, IMAGE_SIZE, DestBitmapWidth*DestBitmapHeight);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "rotateWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t rotateWithCuda(unsigned char *src, unsigned char *dest, unsigned int src_size, unsigned int dest_size)
{
	unsigned char *dev_src = 0;
	unsigned char *dev_dst = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	struct hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);

	cudaStatus = hipMalloc((void**)&dev_src, src_size * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_dst, dest_size * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_src, src, src_size * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	rotateKernel<<<dest_size / 1024, 1024>>>(dev_src, dev_dst, DestBitmapWidth, DestBitmapHeight, sine, cosine, minx, miny);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(dest, dev_dst, dest_size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_src);
	hipFree(dev_dst);
	
	return cudaStatus;
}
